#include "hip/hip_runtime.h"
__device__ float spoc_fadd ( float a, float b ) { return (a + b);}
__device__ float spoc_fminus ( float a, float b ) { return (a - b);}
__device__ float spoc_fmul ( float a, float b ) { return (a * b);}
__device__ float spoc_fdiv ( float a, float b ) { return (a / b);}
__device__ int logical_and (int a, int b ) { return (a & b);}
__device__ int spoc_powint (int a, int b ) { return ((int) pow (((double) a), ((double) b)));}
__device__ int spoc_xor (int a, int b ) { return (a^b);}


__device__ float spoc_fun__1  ( float x ){return sin x;
  }
#ifdef __cplusplus
extern "C" {
#endif

__global__ void spoc_dummy (  float* a, float x ) {
  int i;
  i = blockIdx.x*blockDim.x+threadIdx.x ;
  a[i] = spoc_fun__1 (x) ;
  
}
#ifdef __cplusplus
}
#endif